
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    cuda_hello<<<1,1>>>(); 
    hipDeviceSynchronize();

    print_kernel<<<10, 10>>>();
    hipDeviceSynchronize();
    return 0;
}
